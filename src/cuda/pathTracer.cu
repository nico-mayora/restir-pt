#include "hip/hip_runtime.h"
#include "pathTracer.h"
#include <optix_device.h>

inline __device__ float norm2(const owl::vec2f& v) {
    return v.x * v.x + v.y * v.y;
}

OPTIX_RAYGEN_PROGRAM(ptRayGen)()  {
    const RayGenData &self = owl::getProgramData<RayGenData>();
    const owl::vec2i pixelID = owl::getLaunchIndex();

    PerRayData prd;
    prd.random.init(pixelID.x,pixelID.y);
    owl::vec3f colour = 0.f;

    for (int sampleID=0; sampleID < self.samples; sampleID++) {
        owl::Ray ray;

        const owl::vec2f pixelSample(prd.random(),prd.random());
        const owl::vec2f screen
          = (owl::vec2f(pixelID)+pixelSample)
          / owl::vec2f(self.resolution);
        const owl::vec3f origin = self.camera.pos;
        const owl::vec3f direction
            = normalize(self.camera.dir_00
                + screen.u * self.camera.dir_du
                + screen.v * self.camera.dir_dv);

        ray.origin = origin;
        ray.direction = direction;
        traceRay(self.world, ray, prd);

        colour += prd.colour;
    }

    colour = colour * (1.f / self.samples);

    const int fbOfs = pixelID.x+self.resolution.x*pixelID.y;
    self.fbPtr[fbOfs] = owl::make_rgba(colour);
}


OPTIX_MISS_PROGRAM(miss)()
{
    const owl::vec2i pixelID = owl::getLaunchIndex();
    const MissProgData &self = owl::getProgramData<MissProgData>();

    auto &prd = owl::getPRD<PerRayData>();

    owl::vec3f rayDir = optixGetWorldRayDirection();
    rayDir = normalize(rayDir);
    prd.colour = self.sky_colour * (rayDir.y * .5f + 1.f);
}

OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)()
{
    auto &prd = owl::getPRD<PerRayData>();

    const TrianglesGeomData &self = owl::getProgramData<TrianglesGeomData>();
    const auto colour = self.material->albedo;
    const int   primID = optixGetPrimitiveIndex();
    const owl::vec3f Ng = normalize(self.normal[primID]);

    const owl::vec3f rayDir = optixGetWorldRayDirection();

    prd.colour = (.2f + .8f*fabs(dot(rayDir,Ng))) * colour;
}