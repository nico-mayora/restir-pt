#include "hip/hip_runtime.h"
#include "pathTracer.h"
#include <optix_device.h>

inline __device__ float norm2(const owl::vec2f& v) {
    return v.x * v.x + v.y * v.y;
}

OPTIX_RAYGEN_PROGRAM(ptRayGen)()  {
    const RayGenData &self = owl::getProgramData<RayGenData>();
    const owl::vec2i pixelID = owl::getLaunchIndex();

    const owl::vec2f screen = (owl::vec2f(pixelID)+owl::vec2f(.5f)) / owl::vec2f(self.resolution);
    owl::Ray ray;
    ray.origin = self.camera.pos;
    ray.direction
      = normalize(self.camera.dir_00
                  + screen.u * self.camera.dir_du
                  + screen.v * self.camera.dir_dv);
    if (pixelID == owl::vec2i(600, 400)) {
        printf("----------\npixelID: %d %d\n", pixelID.x, pixelID.y);
        printf("ray org: %f %f %f, ray dir: %f %f %f\n----------\n", ray.origin.x, ray.origin.y, ray.origin.z, ray.direction.x, ray.direction.y, ray.direction.z);
    }

    owl::vec3f color;
    traceRay(self.world, ray, color);

    const int fbOfs = pixelID.x+self.resolution.x*pixelID.y;
    self.fbPtr[fbOfs] = owl::make_rgba(color);
}


OPTIX_MISS_PROGRAM(miss)()
{
    const owl::vec2i pixelID = owl::getLaunchIndex();
    const MissProgData &self = owl::getProgramData<MissProgData>();

    owl::vec3f &prd = owl::getPRD<owl::vec3f>();
    prd = self.sky_colour;
}

OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)()
{
    owl::vec3f &prd = owl::getPRD<owl::vec3f>();

    const TrianglesGeomData &self = owl::getProgramData<TrianglesGeomData>();
    const auto colour = self.material->albedo;
    const int   primID = optixGetPrimitiveIndex();
    const owl::vec3f Ng = normalize(self.normal[primID]);

    const owl::vec3f rayDir = optixGetWorldRayDirection();
    const owl::vec3f rayOrg = optixGetWorldRayOrigin();
    const auto tmax = optixGetRayTmax();
    const owl::vec3f hitpoint = rayOrg + tmax * rayDir;
    if (abs(hitpoint.x - 1.f) < 0.1) {
        prd = owl::vec3f(1.f, 0.f, 0.f);
        return;
    }


    prd = (.2f + .8f*fabs(dot(rayDir,Ng)))*colour;
}